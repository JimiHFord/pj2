
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    OuterProductGpu.cu
// Package: edu.rit.gpu.example
// Unit:    OuterProductGpu kernel function
//
// This C/CUDA source file is copyright (C) 2014 by Alan Kaminsky. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// ark@cs.rit.edu.
//
// This C/CUDA source file is part of the Parallel Java 2 Library ("PJ2"). PJ2
// is free software; you can redistribute it and/or modify it under the terms of
// the GNU General Public License as published by the Free Software Foundation;
// either version 3 of the License, or (at your option) any later version.
//
// PJ2 is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

/**
 * Device kernel to compute the outer product matrix of two vectors.
 * <P>
 * Called with a two-dimensional grid of two-dimensional blocks.
 *
 * @param  a  First vector (input).
 * @param  b  Second vector (input).
 * @param  c  Outer product matrix (output).
 * @param  N  Vector length.
 *
 * @author  Alan Kaminsky
 * @version 13-Apr-2014
 */
extern "C" __global__ void outerProduct
	(double *a,
	 double *b,
	 double **c,
	 int N)
	{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (row < N && col < N)
		c[row][col] = a[row]*b[col];
	}
