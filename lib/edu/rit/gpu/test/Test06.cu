
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    Test06.cu
// Package: edu.rit.gpu.test
// Unit:    Test06 kernel function
//
// This C/CUDA source file is copyright (C) 2014 by Alan Kaminsky. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// ark@cs.rit.edu.
//
// This C/CUDA source file is part of the Parallel Java 2 Library ("PJ2"). PJ2
// is free software; you can redistribute it and/or modify it under the terms of
// the GNU General Public License as published by the Free Software Foundation;
// either version 3 of the License, or (at your option) any later version.
//
// PJ2 is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

/**
 * Compute the sum of two double matrices.
 * <P>
 * Called with a 2-D grid of 2-D blocks. Each thread computes one element of the
 * output matrix.
 *
 * @param  a     First input matrix.
 * @param  b     Second input matrix.
 * @param  c     Output matrix.
 * @param  rows  Number of matrix rows.
 * @param  cols  Number of matrix columns.
 *
 * @author  Alan Kaminsky
 * @version 05-Apr-2014
 */
extern "C" __global__ void addDoubleMatrices
	(double **a,
	 double **b,
	 double **c,
	 int rows,
	 int cols)
	{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (row < rows && col < cols)
		c[row][col] = a[row][col] + b[row][col];
	}
