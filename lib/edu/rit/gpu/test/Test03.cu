
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    Test03.cu
// Package: edu.rit.gpu.test
// Unit:    Test03 kernel function
//
// This C/CUDA source file is copyright (C) 2014 by Alan Kaminsky. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// ark@cs.rit.edu.
//
// This C/CUDA source file is part of the Parallel Java 2 Library ("PJ2"). PJ2
// is free software; you can redistribute it and/or modify it under the terms of
// the GNU General Public License as published by the Free Software Foundation;
// either version 3 of the License, or (at your option) any later version.
//
// PJ2 is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

/**
 * Compute the sum of two long vectors.
 * <P>
 * Called with a 1-D grid of 1-D blocks. Each thread computes one element of the
 * output vector.
 *
 * @param  a    First input vector.
 * @param  b    Second input vector.
 * @param  c    Output vector.
 * @param  len  Vector length.
 *
 * @author  Alan Kaminsky
 * @version 20-Feb-2014
 */
extern "C" __global__ void addLongVectors
	(long long *a,
	 long long *b,
	 long long *c,
	 int len)
	{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < len)
		c[i] = a[i] + b[i];
	}
